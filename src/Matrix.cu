#include "hip/hip_runtime.h"
#include "Math/Matrix.hpp"
#include <iostream>
#define BLOCK_DIM 32

namespace math {

    template <typename T>
    void Matrix<T>::init(int rows, int cols) {
        int rowsPadded = rows;
        int colsPadded = cols;
        if (rows % BLOCK_DIM != 0) {
            rowsPadded += BLOCK_DIM - (rows % BLOCK_DIM);
        }
        if (cols % BLOCK_DIM != 0) {
            colsPadded += BLOCK_DIM - (cols % BLOCK_DIM);
        }
        elements = std::vector<T> (rowsPadded * colsPadded);
        this -> rowsRaw = rowsPadded;
        this -> colsRaw = colsPadded;
        this -> rows = rows;
        this -> cols = cols;
    }

    template <typename T>
    Matrix<T>::Matrix() {
        // Initialize elements with size (rowsRaw, colsRaw).
        init(0, 0);
    }

    template <typename T>
    Matrix<T>::Matrix(int rows, int cols) {
        // Initialize elements with size (rowsRaw, colsRaw).
        init(rows, cols);
    }

    template <typename T>
    Matrix<T>::Matrix(const std::vector<T>& initialElements, int rows, int cols) {
        // Initialize elements with size (rowsRaw, colsRaw).
        init(rows, cols);
        for (int i = 0; i < size(); ++i) {
            at(i) = initialElements.at(i);
        }
    }

    template <typename T>
    Matrix<T>::Matrix(const std::vector<std::vector<T> >& initialElements) {
        this -> rows = initialElements.size();
        this -> cols = initialElements.at(0).size();
        init(rows, cols);
        for (int row = 0; row < rows; ++row) {
            for (int col = 0; col < cols; ++col) {
                at(row, col) = initialElements.at(row).at(col);
            }
        }
    }

    template <typename T>
    T& Matrix<T>::at(int row, int col) {
        return elements.at(row * numColumnsRaw() + col);
    }

    template <typename T>
    const T& Matrix<T>::at(int row, int col) const {
        return elements.at(row * numColumnsRaw() + col);
    }

    template <typename T>
    T& Matrix<T>::at(int index) {
        int row = index / numColumns();
        int col = index % numColumns();
        return at(row, col);
    }

    template <typename T>
    const T& Matrix<T>::at(int index) const {
        return at(index / numColumns(), index % numColumns());
    }

    template <typename T>
    T* Matrix<T>::data() {
        return elements.data();
    }

    template <typename T>
    const T* Matrix<T>::data() const {
        return elements.data();
    }

    template <typename T>
    std::vector<T>& Matrix<T>::raw() {
        return elements;
    }

    template <typename T>
    const std::vector<T>& Matrix<T>::raw() const {
        return elements;
    }

    template <typename T>
    std::vector<T> Matrix<T>::getElements() const {
        std::vector<T> temp;
        std::vector<T> tempRow;
        temp.reserve(size());
        for (int i = 0; i < numRows(); ++i) {
            tempRow = row(i);
            temp.insert(temp .end(), tempRow.cbegin(), tempRow.cend());
        }
        return temp;
    }

    template <typename T>
    int Matrix<T>::numRowsRaw() const {
        return rowsRaw;
    }

    template <typename T>
    int Matrix<T>::numColumnsRaw() const {
        return colsRaw;
    }

    template <typename T>
    int Matrix<T>::numRows() const {
        return rows;
    }

    template <typename T>
    int Matrix<T>::numColumns() const {
        return cols;
    }

    template <typename T>
    int Matrix<T>::sizeRaw() const {
        return numColumnsRaw() * numRowsRaw();
    }

    template <typename T>
    int Matrix<T>::size() const {
        return numColumns() * numRows();
    }

    template <typename T>
    std::vector<T> Matrix<T>::row(int row) const {
        std::vector<T> tempRow;
        tempRow.reserve(numColumns());
        for (int i = 0; i < numColumns(); ++i) {
            tempRow.push_back(at(row, i));
        }
        return tempRow;
    }

    template <typename T>
    std::vector<T> Matrix<T>::column(int col) const {
        std::vector<T> tempCol;
        tempCol.reserve(numRows());
        for (int i = 0; i < numRows(); ++i) {
            tempCol.push_back(at(i, col));
        }
        return tempCol;
    }

    template<typename T>
    __global__ void computeTranspose(T* original, int numRows, int numCols, T* transposed) {
        // Avoid bank conflicts by allocating a single dummy element.
        __shared__ T tile[BLOCK_DIM][BLOCK_DIM + 1];
        // Compute row and column of this block.
        int row = blockIdx.x * BLOCK_DIM;
        int col = blockIdx.y * BLOCK_DIM;
        // Load a (transposed) tile into shared memory.
        tile[threadIdx.y][threadIdx.x] = original[(row + threadIdx.x) * numCols + (col + threadIdx.y)];
        // Synchronize.
        __syncthreads();
        // Write the tiles into the output. Switch rows and columns to handle non-square matrices.
        transposed[(col + threadIdx.x) * numRows + (row + threadIdx.y)] = tile[threadIdx.x][threadIdx.y];
    }

    template <typename T>
    Matrix<T> Matrix<T>::transpose() const {
        int size = sizeRaw();
        Matrix<T> transpose = Matrix<T>(numColumns(), numRows());
        // Initialize device copies.
        T *dev_original, *dev_transposed;
        // Allocate memory for device ccpies.
        hipMalloc((void**)&dev_original, size * sizeof(T));
        hipMalloc((void**)&dev_transposed, size * sizeof(T));
        // Copy inputs to device.
        hipMemcpy(dev_original, data(), size * sizeof(T), hipMemcpyHostToDevice);
        // Launch kernel with only as many blocks as necessary.
        dim3 blocks(numRowsRaw() / BLOCK_DIM, numColumnsRaw() / BLOCK_DIM);
        dim3 threads(BLOCK_DIM, BLOCK_DIM);
        computeTranspose<<<blocks, threads>>>(dev_original, numRowsRaw(), numColumnsRaw(), dev_transposed);
        // Get result.
        hipMemcpy(transpose.data(), dev_transposed, size * sizeof(T) , hipMemcpyDeviceToHost);
        // Free memory.
        hipFree(dev_original);
        hipFree(dev_transposed);
        // Return.
        return transpose;
    }

    template <typename T>
    __global__ void computeProduct(T* A, T* B, int numColsA, int numColsB, T* C) {
        // Avoid bank conflicts by allocating a single dummy element.
        __shared__ T tileA[BLOCK_DIM][BLOCK_DIM + 1];
        __shared__ T tileB[BLOCK_DIM][BLOCK_DIM + 1];
        // Compute the coordinates of matrix C that this thread is responsible for.
        int row = blockIdx.x * BLOCK_DIM + threadIdx.x;
        int col = blockIdx.y * BLOCK_DIM + threadIdx.y;
        T Cvalue = T();
        // Iterate over the sub-matrices of A and B.
        for (int i = 0; i < numColsA; i += BLOCK_DIM) {
            // Load sub-matrix A.
            tileA[threadIdx.x][threadIdx.y] = A[row * numColsA + (i + threadIdx.y)];
            // Load sub-matrix B.
            tileB[threadIdx.x][threadIdx.y] = B[(i + threadIdx.x) * numColsB + col];
            // Synchronize.
            __syncthreads();
            // Compute dot product only if the point is within the C matrix.
            #pragma unroll
            for (int j = 0; j < BLOCK_DIM; ++j) {
                Cvalue += tileA[threadIdx.x][j] * tileB[j][threadIdx.y];
            }
            // Synchronize.
            __syncthreads();
        }
        // Write to output.
        C[row * numColsB + col] = Cvalue;
    }

    template <typename T>
    Matrix<T> Matrix<T>::operator*(const Matrix<T>& other) const {
        if (numColumns() != other.numRows()) {
            throw std::invalid_argument("Incompatible matrices cannot be multiplied.");
        }
        Matrix product = Matrix(numRows(), other.numColumns());
        int Asize = sizeRaw();
        int Bsize = other.sizeRaw();
        int Csize = product.sizeRaw();
        // Initialize device copies.
        T *dev_A, *dev_B, *dev_C;
        // Allocate memory for device ccpies.
        hipMalloc((void**)&dev_A, Asize * sizeof(T));
        hipMalloc((void**)&dev_B, Bsize * sizeof(T));
        hipMalloc((void**)&dev_C, Csize * sizeof(T));
        // Copy inputs to device.
        hipMemcpy(dev_A, data(), Asize * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(dev_B, other.data(), Bsize * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(dev_C, product.data(), Csize * sizeof(T), hipMemcpyHostToDevice);
        // Launch kernel with only as many blocks as necessary.
        dim3 blocks(product.numRowsRaw() / BLOCK_DIM, product.numColumnsRaw() / BLOCK_DIM);
        dim3 threads(BLOCK_DIM, BLOCK_DIM);
        computeProduct<<<blocks, threads>>>(dev_A, dev_B, numColumnsRaw(), other.numColumnsRaw(), dev_C);
        // Get result.
        hipMemcpy(product.data(), dev_C, Csize * sizeof(T) , hipMemcpyDeviceToHost);
        // Free memory.
        hipFree(dev_A);
        hipFree(dev_B);
        hipFree(dev_C);
        // Return.
        return product;
    }

    template <typename T>
    __global__ void computeScalarProduct(T* A, T B, T* C) {
        C[blockIdx.x * BLOCK_DIM + threadIdx.x] = A[blockIdx.x * BLOCK_DIM + threadIdx.x] * B;
    }

    template <typename T>
    Matrix<T> Matrix<T>::operator*(T other) const {
        Matrix product = Matrix(numRows(), numColumns());
        int size = sizeRaw();
        // Initialize device copies.
        T *dev_A, *dev_C;
        // Allocate memory for device copies.
        hipMalloc((void**)&dev_A, size * sizeof(T));
        hipMalloc((void**)&dev_C, size * sizeof(T));
        // Copy inputs to device.
        hipMemcpy(dev_A, data(), size * sizeof(T), hipMemcpyHostToDevice);
        // Launch kernel where numThreads = size of matrix.
        dim3 blocks(sizeRaw() / BLOCK_DIM);
        dim3 threads(BLOCK_DIM);
        computeScalarProduct<<<blocks, threads>>>(dev_A, other, dev_C);
        // Get result.
        hipMemcpy(product.data(), dev_C, size * sizeof(T) , hipMemcpyDeviceToHost);
        // Free memory.
        hipFree(dev_A);
        hipFree(dev_C);
        // Return.
        return product;
    }

    template <typename T>
    __global__ void computeSum(T* A, T* B, T* C) {
        C[blockIdx.x * BLOCK_DIM + threadIdx.x] = A[blockIdx.x * BLOCK_DIM + threadIdx.x] + B[blockIdx.x * BLOCK_DIM + threadIdx.x];
    }

    template <typename T>
    Matrix<T> Matrix<T>::operator+(const Matrix<T>& other) const {
        if (size() != other.size()) {
            throw std::invalid_argument("Incompatible matrices cannot be added.");
        }
        Matrix sum = Matrix(numRows(), numColumns());
        int size = sizeRaw();
        // Initialize device copies.
        T *dev_A, *dev_B, *dev_C;
        // Allocate memory for device copies.
        hipMalloc((void**)&dev_A, size * sizeof(T));
        hipMalloc((void**)&dev_B, size * sizeof(T));
        hipMalloc((void**)&dev_C, size * sizeof(T));
        // Copy inputs to device.
        hipMemcpy(dev_A, data(), size * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(dev_B, other.data(), size * sizeof(T), hipMemcpyHostToDevice);
        // Launch kernel where numThreads = size of matrix.
        dim3 blocks(sizeRaw() / BLOCK_DIM);
        dim3 threads(BLOCK_DIM);
        computeSum<<<blocks, threads>>>(dev_A, dev_B, dev_C);
        // Get result.
        hipMemcpy(sum.data(), dev_C, size * sizeof(T) , hipMemcpyDeviceToHost);
        // Free memory.
        hipFree(dev_A);
        hipFree(dev_B);
        hipFree(dev_C);
        // Return.
        return sum;
    }

    template <typename T>
    __global__ void computeDifference(T* A, T* B, T* C) {
        C[blockIdx.x * BLOCK_DIM + threadIdx.x] = A[blockIdx.x * BLOCK_DIM + threadIdx.x] - B[blockIdx.x * BLOCK_DIM + threadIdx.x];
    }

    template <typename T>
    Matrix<T> Matrix<T>::operator-(const Matrix<T>& other) const {
        if (size() != other.size()) {
            throw std::invalid_argument("Incompatible matrices cannot be added.");
        }
        Matrix sum = Matrix(numRows(), numColumns());
        int size = sizeRaw();
        // Initialize device copies.
        T *dev_A, *dev_B, *dev_C;
        // Allocate memory for device copies.
        hipMalloc((void**)&dev_A, size * sizeof(T));
        hipMalloc((void**)&dev_B, size * sizeof(T));
        hipMalloc((void**)&dev_C, size * sizeof(T));
        // Copy inputs to device.
        hipMemcpy(dev_A, data(), size * sizeof(T), hipMemcpyHostToDevice);
        hipMemcpy(dev_B, other.data(), size * sizeof(T), hipMemcpyHostToDevice);
        // Launch kernel where numThreads = size of matrix.
        dim3 blocks(sizeRaw() / BLOCK_DIM);
        dim3 threads(BLOCK_DIM);
        computeDifference<<<blocks, threads>>>(dev_A, dev_B, dev_C);
        // Get result.
        hipMemcpy(sum.data(), dev_C, size * sizeof(T) , hipMemcpyDeviceToHost);
        // Free memory.
        hipFree(dev_A);
        hipFree(dev_B);
        hipFree(dev_C);
        // Return.
        return sum;
    }

    template class Matrix<int>;
    template class Matrix<float>;
    template class Matrix<double>;
}
