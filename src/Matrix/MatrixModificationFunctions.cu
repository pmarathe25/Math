#ifndef MATRIX_MODIFICATION_FUNCTIONS
#define MATRIX_MODIFICATION_FUNCTIONS

namespace math {
    template <typename T>
    void Matrix<T>::reshape(int rows, int cols) {
        if (rows * cols == size()) {
            this -> rows = rows;
            this -> cols = cols;
            this -> isVec = (rows == 1) || (cols == 1);
        } else {
            throw std::invalid_argument("Size mismatch in reshape.");
        }
    }

    template <typename T>
    void Matrix<T>::set(T setValue) {
        dim3 blocks(ceilDivide(size(), THREADS_PER_BLOCK));
        dim3 threads(THREADS_PER_BLOCK);
        setCUDA<<<blocks, threads>>>(data(), setValue, size());
        hipDeviceSynchronize();
    }

    template <typename T>
    Matrix<T> Matrix<T>::randomNormal(int rows, int cols, double mean, double stdDev) {
        Matrix<T> output(rows, cols);
        auto value = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch());
        std::default_random_engine generator(value.count());
        std::normal_distribution<double> normalDistribution(mean, stdDev);
        for (int i = 0; i < output.size(); ++i) {
            output[i] = normalDistribution(generator);
        }
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::randomNormalLike(const Matrix& like, double mean, double stdDev) {
        Matrix<T> output(like.numRows(), like.numColumns());
        auto value = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch());
        std::default_random_engine generator(value.count());
        std::normal_distribution<double> normalDistribution(mean, stdDev);
        for (int i = 0; i < output.size(); ++i) {
            output[i] = normalDistribution(generator);
        }
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::randomUniform(int rows, int cols, double lowerBound, double upperBound) {
        Matrix<T> output(rows, cols);
        auto value = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch());
        std::default_random_engine generator(value.count());
        std::uniform_real_distribution<double> uniformDistribution(lowerBound, upperBound);
        for (int i = 0; i < output.size(); ++i) {
            output[i] = uniformDistribution(generator);
        }
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::randomUniformLike(const Matrix& like, double lowerBound, double upperBound) {
        Matrix<T> output(like.numRows(), like.numColumns());
        auto value = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now().time_since_epoch());
        std::default_random_engine generator(value.count());
        std::uniform_real_distribution<double> uniformDistribution(lowerBound, upperBound);
        for (int i = 0; i < output.size(); ++i) {
            output[i] = uniformDistribution(generator);
        }
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::ones(int rows, int cols) {
        Matrix<T> output(rows, cols);
        output.set(1);
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::onesLike(const Matrix& like) {
        Matrix<T> output(like.numRows(), like.numColumns());
        output.set(1);
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::zeros(int rows, int cols) {
        Matrix<T> output(rows, cols);
        output.set(0);
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::zerosLike(const Matrix& like) {
        Matrix<T> output(like.numRows(), like.numColumns());
        output.set(0);
        return output;
    }

    template <typename T>
    Matrix<T> Matrix<T>::sequentialMatrix(int rows, int cols) {
        Matrix<T> output(rows, cols);
        for (int i = 0; i < output.size(); ++i) {
            output[i] = i;
        }
        return output;
    }
}

#endif
