#include "hip/hip_runtime.h"
#ifndef MATRIX_CUDA_FUNCTIONS
#define MATRIX_CUDA_FUNCTIONS

namespace math {
    template <typename T>
    __global__ void computeTranspose(const T* original, int numRows, int numCols, T* transposed) {
        int x = blockIdx.y * BLOCK_DIM + threadIdx.x;
        int y = blockIdx.x * BLOCK_DIM + threadIdx.y;
        if (x < numCols && y < numRows) {
            transposed[x * numRows + y] = original[y * numCols + x];
        }
    }

    template <typename T>
    __global__ void computeRowMean(const T* A, float scaleFactor, int numCols, int size, T* C) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        float mean = 0;
        if (col < numCols) {
            for (int i = col; i < size; i += numCols) {
                mean += A[i] * scaleFactor;
            }
            C[col] = mean;
        }
    }

    template <typename T>
    __global__ void computeRowWiseDotProduct(const T* A, const T* B, int numRows, int numCols, T* C) {
        // Each thread computes one row.
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        if (row < numRows) {
            T Cvalue = T();
            // numCols is guaranteed to be a multiple of 32.
            for (int i = 0; i < numCols; ++i) {
                Cvalue += A[i] * B[i];
            }
            C[row] = Cvalue;
        }
    }

    template <typename T>
    __global__ void computeProduct(const T* A, const T* B, int numRowsA, int numColsA, int numColsB, int Asize, int Bsize, T* C) {
       __shared__ T tileA[BLOCK_DIM][BLOCK_DIM + 1];
       __shared__ T tileB[BLOCK_DIM][BLOCK_DIM + 1];
       // Compute the coordinates of matrix C that this thread is responsible for.
       int row = blockIdx.x * blockDim.x + threadIdx.x;
       int col = blockIdx.y * blockDim.y + threadIdx.y;
       bool cValid = row < numRowsA && col < numColsB;
       T Cvalue = T();
       // Iterate over the sub-matrices of A and B.
       int maxIterations = numColsA + BLOCK_DIM - 1;
       for (int i = 0; i < maxIterations; i += BLOCK_DIM) {
           // Compute indices.
           int indexA = row * numColsA + (i + threadIdx.y);
           int indexB = (i + threadIdx.x) * numColsB + col;
           // Load sub-matrix A.
           tileA[threadIdx.x][threadIdx.y] = (indexA < Asize) ? A[indexA] : 0;
           // Load sub-matrix B.
           tileB[threadIdx.x][threadIdx.y] = (indexB < Bsize) ? B[indexB] : 0;
           // Synchronize.
           __syncthreads();
           // Compute dot product only if the point is within the C matrix.
           if (cValid) {
               #pragma unroll
               for (int j = 0; j < BLOCK_DIM; ++j) {
                   Cvalue += tileA[threadIdx.x][j] * tileB[j][threadIdx.y];
               }
           }
           // Synchronize.
           __syncthreads();
       }
       // Write to output.
       if (cValid) {
           C[row * numColsB + col] = Cvalue;
       }
    }

    template <typename T>
    __global__ void computeSum(const T* A, const T* B, int Asize, T* C) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < Asize) {
            C[index] = A[index] + B[index];
        }
    }

    template <typename T>
    __global__ void computeDifference(const T* A, const T* B, int size, T* C) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < size) {
            C[index] = A[index] - B[index];
        }
    }

    template <typename T>
    __global__ void computeMatrixVectorRowSum(const T* A, const T* B, int numCols, int numRowsA, T* C) {
        __shared__ T tileB[BLOCK_DIM];
        // Compute the coordinates of matrix C that this thread is responsible for.
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        int col = blockIdx.y * blockDim.y + threadIdx.y;
        // Load vector - only load each element once.
        if (threadIdx.x == 0) {
            tileB[threadIdx.y] = (col < numCols) ? B[col] : 0;
        }
        // Synchronize.
        __syncthreads();
        // Write to output.
        if (row < numRowsA && col < numCols) {
            int index = row * numCols + col;
            C[index] = A[index] + tileB[threadIdx.y];
        }
    }

    template <typename T>
    __global__ void computeMatrixVectorColumnSum(const T* A, const T* B, int numRows, int numColsA, T* C) {
        __shared__ T tileB[BLOCK_DIM];
        // Compute the coordinates of matrix C that this thread is responsible for.
        int row = blockIdx.x * blockDim.x + threadIdx.x;
        int col = blockIdx.y * blockDim.y + threadIdx.y;
        // Load vector - only load each element once.
        if (threadIdx.y == 0) {
            tileB[threadIdx.x] = (row < numRows) ? B[row] : 0;
        }
        // Synchronize.
        __syncthreads();
        // Write to output.
        if (row < numRows && col < numColsA) {
            int index = row * numColsA + col ;
            C[index] = A[index] + tileB[threadIdx.x];
        }
    }

    template <typename T>
    __global__ void computeScalarProduct(const T* A, T B, int Asize, T* C) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < Asize) {
            C[index] = A[index] * B;
        }
    }

    template <typename T>
    __global__ void computeScalarSum(const T* A, T B, int Asize, T* C) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < Asize) {
            C[index] = A[index] + B;
        }
    }

    template <typename T>
    __global__ void computeHadamardProduct(const T* A, const T* B, int Asize, T* C) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index < Asize) {
            C[index] = A[index] * B[index];
        }
    }
}

#endif
