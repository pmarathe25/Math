#include "hip/hip_runtime.h"
#include "Matrix.hpp"
#include <string>
#include <iostream>
#include <chrono>

int testMatrixCreation() {
    std::cout << "========================Testing Matrix Creation.========================" << '\n';
    std::cout << "Vector of Vectors Creation 2x3" << '\n';
    Matrix_F creationTest0 = Matrix_F({{0, 1, 0}, {0, 2, 3}});
    creationTest0.display();
    std::cout << "Vector Creation 2x3" << '\n';
    Matrix_F creationTest1 = Matrix_F({0, 1, 2, 3, 4, 5}, 2, 3);
    creationTest1.display();
    std::cout << "Empty Initialization with Assignment 40x40" << '\n';
    Matrix_D creationTest2 = Matrix_D::ones(40, 40);
    creationTest2.display();
    std::cout << "Empty Initialization with Assignment 5x30" << '\n';
    Matrix_F creationTest3 = Matrix_F::sequentialMatrix(5, 30);
    creationTest3.display();
    std::cout << "Empty Initialization with Assignment 15x5" << '\n';
    Matrix_F creationTest4 = Matrix_F::sequentialMatrix(15, 5);
    creationTest4.display();
    return 0;
}

int testMatrixCopy() {
    std::cout << "========================Testing Matrix Copy========================" << '\n';
    std::cout << "Copying Matrices of the same type" << '\n';
    Matrix_F copyTest0 = Matrix_F::sequentialMatrix(2, 10);
    Matrix_F copyTest1 = copyTest0;
    std::cout << "Original" << '\n';
    copyTest0.display();
    std::cout << "Copy" << '\n';
    copyTest1.display();
    std::cout << "Modifying First Matrix (Should not affect second)" << '\n';
    copyTest0.at(0) = 4056;
    std::cout << "Original" << '\n';
    copyTest0.display();
    std::cout << "Copy" << '\n';
    copyTest1.display();
    return 0;
}

int testMatrixReshape() {
    std::cout << "========================Testing Matrix Reshape========================" << '\n';
    Matrix_F reshapeTest0 = Matrix_F::sequentialMatrix(5, 30);
    reshapeTest0.display();
    std::cout << "Reshaping 5x30 Matrix into 10x15" << '\n';
    reshapeTest0.reshape(10, 15);
    reshapeTest0.display();
    return 0;
}

int testMatrixTranspose() {
    std::cout << "========================Testing Matrix Transpose========================" << '\n';
    Matrix_F transposeTest0 = Matrix_F::sequentialMatrix(5, 30);
    transposeTest0.display();
    std::cout << "Transpose" << '\n';
    transposeTest0.transpose().display();
    return 0;
}

int testMatrixRandomize() {
    std::cout << "========================Testing Matrix Randomize========================" << '\n';
    std::cout << "Normal Randomization | Mean 0 | Standard Deviation 1" << '\n';
    Matrix_F randomizeTest0 = Matrix_F::sequentialMatrix(5, 30);
    randomizeTest0 = Matrix_F::randomNormalLike(randomizeTest0, 0.0, 1.0);
    randomizeTest0.display();
    std::cout << "Uniform Randomization | Range [0, 1]" << '\n';
    randomizeTest0 = Matrix_F::randomUniformLike(randomizeTest0, 0.0, 1.0);
    randomizeTest0.display();
    return 0;
}

int testRowMean() {
    std::cout << "========================Testing Matrix Row Mean========================" << '\n';
    std::cout << "5x30 Sequential Matrix" << '\n';
    Matrix rowMeanTest0 = Matrix::sequentialMatrix(5, 30);
    rowMeanTest0.display();
    std::cout << "Row Mean" << '\n';
    rowMeanTest0.rowMean().display();
    return 0;
}

int testDotProduct() {
    std::cout << "========================Testing Row-Wise Dot Product========================" << '\n';
    std::cout << "30x5 Ones Matrix" << '\n';
    Matrix dotTest0 = Matrix::ones(30, 5);
    dotTest0.display();
    std::cout << "Row-Wise Dot Product of Matrix with itself" << '\n';
    dotTest0.dot(dotTest0).display();
    return 0;
}

int testMatrixMultiplication() {
    std::cout << "========================Testing Matrix Multiplication========================" << '\n';
    std::cout << "10x5 Ones Matrix" << '\n';
    Matrix multiplicationTest0 = Matrix::ones(10, 5);
    multiplicationTest0.display();
    std::cout << "5x5 Sequential Matrix" << '\n';
    Matrix multiplicationTest1 = Matrix::sequentialMatrix(5, 5);
    multiplicationTest1.display();
    std::cout << "Product" << '\n';
    (multiplicationTest0 * multiplicationTest1).display();
    return 0;
}

int testMatrixArithmetic() {
    std::cout << "========================Testing Matrix-Matrix Arithmetic========================" << '\n';
    std::cout << "10x10 Ones Matrix" << '\n';
    Matrix_F arithmeticTest0 = Matrix_F::ones(10, 10);
    arithmeticTest0.display();
    std::cout << "10x10 Sequential Matrix" << '\n';
    Matrix_F arithmeticTest1 = Matrix_F::sequentialMatrix(10, 10);
    arithmeticTest1.display();
    std::cout << "Sum" << '\n';
    Matrix_F sum = arithmeticTest0 + arithmeticTest1;
    sum.display();
    std::cout << "Difference" << '\n';
    Matrix_F difference = arithmeticTest0 - arithmeticTest1;
    difference.display();
    std::cout << "Sum in place" << '\n';
    arithmeticTest0 += arithmeticTest1;
    arithmeticTest0.display();
    std::cout << "Difference in place" << '\n';
    arithmeticTest0 -= arithmeticTest1;
    arithmeticTest0.display();
    return 0;
}

int testMatrixVectorArithmetic() {
    std::cout << "========================Testing Matrix-Vector Arithmetic========================" << '\n';
    std::cout << "10x10 Ones Matrix" << '\n';
    Matrix vectorArithmeticTest0 = Matrix::ones(10, 10);
    vectorArithmeticTest0.display();
    std::cout << "1x10 Sequential Column Vector" << '\n';
    Matrix vectorArithmeticTest1 = Matrix::sequentialMatrix(10, 1);
    vectorArithmeticTest1.display();
    std::cout << "Matrix-Column Vector Addition" << '\n';
    vectorArithmeticTest0.addVector(vectorArithmeticTest1).display();
    std::cout << "1x10 Sequential Row Vector" << '\n';
    Matrix vectorArithmeticTest2 = Matrix::sequentialMatrix(1, 10);
    vectorArithmeticTest2.display();
    std::cout << "Matrix-Row Vector Addition" << '\n';
    vectorArithmeticTest0.addVector(vectorArithmeticTest2).display();
    return 0;
}

int testMatrixScalarArithmetic() {
    std::cout << "========================Testing Matrix-Scalar Arithmetic========================" << '\n';
    std::cout << "10x10 Ones Matrix" << '\n';
    Matrix_F scalarArithmeticTest0 = Matrix_F::ones(10, 10);
    scalarArithmeticTest0.display();
    std::cout << "Division with Scalar 2 (Right)" << '\n';
    (scalarArithmeticTest0 / 2).display();
    std::cout << "Product with Scalar 2 (Right)" << '\n';
    (scalarArithmeticTest0 * 2).display();
    std::cout << "Product with Scalar 2 (Left)" << '\n';
    (2 * scalarArithmeticTest0).display();
    std::cout << "Sum with Scalar 2 (Right)" << '\n';
    (scalarArithmeticTest0 + 2).display();
    std::cout << "Sum with Scalar 2 (Left)" << '\n';
    (2 + scalarArithmeticTest0).display();
    std::cout << "Difference with Scalar 2 (Right)" << '\n';
    (scalarArithmeticTest0 - 2).display();
    std::cout << "Difference with Scalar 2 (Left)" << '\n';
    (2 - scalarArithmeticTest0).display();
    return 0;
}

int testMatrixHadamardProduct() {
    std::cout << "========================Testing Matrix Hadamard Product========================" << '\n';
    std::cout << "10x10 Sequential Matrix" << '\n';
    Matrix hadamardProductTest0 = Matrix::sequentialMatrix(10, 10);
    hadamardProductTest0.display();
    std::cout << "Matrix Hadamard Product with itself" << '\n';
    hadamardProductTest0.hadamard(hadamardProductTest0).display();
    return 0;
}

__device__ double sigmoid(double a) {
    return 1 / (1 + exp(-a));
}

int testMatrixApplyFunction() {
    std::cout << "========================Testing Matrix Apply Function========================" << '\n';
    std::cout << "10x10 Sequential Matrix" << '\n';
    Matrix_D applyFunctionTest0 = Matrix_D::sequentialMatrix(10, 10) - 50;
    applyFunctionTest0.display();
    std::cout << "Applying sigmoid function" << '\n';
    applyFunctionTest0.applyFunction<sigmoid>().display();
    return 0;
}

int testMatrixPower() {
    std::cout << "========================Testing Matrix Power========================" << '\n';
    std::cout << "10x10 Sequential Matrix" << '\n';
    Matrix_D powerTest0 = Matrix_D::sequentialMatrix(10, 10);
    powerTest0.display();
    std::cout << "Computing square" << '\n';
    math::pow(powerTest0, 2).display();
    std::cout << "Computing cube" << '\n';
    math::pow(powerTest0, 3).display();
    return 0;
}

int testMatrixFileIO() {
    std::string filePath = "./test/matrix";
    std::cout << "========================Testing Matrix File IO========================" << '\n';
    std::cout << "100x100 Random Normal Matrix" << '\n';
    Matrix_D fileIOTest0 = Matrix_D::randomNormal(6, 8);
    fileIOTest0.display();
    std::cout << "Writing matrix" << '\n';
    fileIOTest0.save(filePath);
    std::cout << "Loading matrix" << '\n';
    Matrix_D fileIOTest1(filePath);
    std::cout << "Loaded matrix of dimensions " << fileIOTest1.numRows() << "x" << fileIOTest1.numColumns() << '\n';
    fileIOTest1.display();
    std::cout << '\n';
    return 0;
}

int main() {
    int numFailed = 0;
    numFailed += testMatrixCreation();
    numFailed += testMatrixCopy();
    numFailed += testMatrixReshape();
    numFailed += testMatrixTranspose();
    numFailed += testMatrixRandomize();
    numFailed += testRowMean();
    numFailed += testDotProduct();
    numFailed += testMatrixMultiplication();
    numFailed += testMatrixArithmetic();
    numFailed += testMatrixVectorArithmetic();
    numFailed += testMatrixScalarArithmetic();
    numFailed += testMatrixHadamardProduct();
    numFailed += testMatrixApplyFunction();
    numFailed += testMatrixPower();
    numFailed += testMatrixFileIO();
    if (numFailed == 0) {
        std::cout << "All Tests Passed." << '\n';
    } else {
        std::cout << numFailed << " Tests Failed." << '\n';
    }
}
